#include<stdio.h>
#include<hip/hip_runtime.h>

__global__ void bcast(int arg) {
    int laneId = threadIdx.x & 0x1f; //转换线程ID为[0, 31]
    int value;
    if(laneId == 0) {
        value = arg;
    }
    value = __shfl_sync(0xfffffff, value, 0, 32);
    if(value != arg) {
        printf("Thread %d failed.\n", threadIdx.x);
    }
}

int main() {
        bcast<<<1, 32>>>(1234);
        hipDeviceSynchronize();

        return 0;
}