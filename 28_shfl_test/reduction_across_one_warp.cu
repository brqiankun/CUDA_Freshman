#include<stdio.h>
#include<hip/hip_runtime.h>

__global__ void warpReduce() {
    int laneId = threadIdx.x & 0x1f;
    int value = 8 - laneId;

    for(int i = 4; i >= 1; i/=2) {
        value += __shfl_xor_sync(0xffffffff, value, i, 32);
        printf("thread %d  i=%d value = %d\n", threadIdx.x, i, value);
        __syncthreads();
    }

    printf("thread %d final value = %d\n", threadIdx.x, value);
}

int main() {
    warpReduce<<<1, 8>>>();
    hipDeviceSynchronize();
}
